#include "hits.hxx"

using namespace gunrock;
using namespace memory;

void test_hits(int argc, char** argv) {
  if (2 != argc) {
    std::cerr << "usage:: ./bin/<program-name> filename.mtx \n";
    exit(1);
  }

  using vertex_t = int;
  using edge_t = int;
  using weight_t = float;
  using csr_t =
      format::csr_t<memory_space_t::device, vertex_t, edge_t, weight_t>;

  csr_t csr;

  std::string filename = argv[1];
  if (util::is_market(filename)) {
    io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
    csr.from_coo(mm.load(filename));
  } else if (util::is_binary_csr(filename)) {
    csr.read_binary(filename);
  } else {
    std::cerr << "Unknown file format: " << filename << std::endl;
    exit(1);
  }

  auto G = graph::build::from_csr<memory_space_t::device, graph::view_t::csr>(
      csr.number_of_rows,               // rows
      csr.number_of_columns,            // columns
      csr.number_of_nonzeros,           // nonzeros
      csr.row_offsets.data().get(),     // row_offsets
      csr.column_indices.data().get(),  // column_indices
      csr.nonzero_values.data().get()   // values
  );

  hits::param_c param{20};
  hits::result_c result{G};

  auto time = gunrock::hits::run(G, param, result);
  // result.print_result(20);
  std::cout << "GPU Elapsed Time : " << time << " (ms)" << std::endl;
}

int main(int argc, char** argv) {
  test_hits(argc, argv);
  return 0;
}
