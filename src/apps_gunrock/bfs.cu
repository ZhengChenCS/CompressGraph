#include <gunrock/algorithms/bfs.hxx>
#include "bfs_cpu.hxx"  // Reference implementation

using namespace gunrock;
using namespace memory;

void test_bfs(int num_arguments, char** argument_array) {
  if (num_arguments != 2) {
    std::cerr << "usage: ./bin/<program-name> filename.mtx" << std::endl;
    exit(1);
  }

  // --
  // Define types

  using vertex_t = int;
  using edge_t = int;
  using weight_t = float;

  using csr_t =
      format::csr_t<memory_space_t::device, vertex_t, edge_t, weight_t>;

  // --
  // IO

  csr_t csr;
  std::string filename = argument_array[1];

  if (util::is_market(filename)) {
    io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
    std::cout << "here" << std::endl;
    csr.from_coo(mm.load(filename));
    std::cout << "here" << std::endl;
  } else if (util::is_binary_csr(filename)) {
    csr.read_binary(filename);
  } else {
    std::cerr << "Unknown file format: " << filename << std::endl;
    exit(1);
  }

  thrust::device_vector<vertex_t> row_indices(csr.number_of_nonzeros);
  thrust::device_vector<vertex_t> column_indices(csr.number_of_nonzeros);
  thrust::device_vector<edge_t> column_offsets(csr.number_of_columns + 1);

  // --
  // Build graph + metadata

  auto G =
      graph::build::from_csr<memory_space_t::device,
                             graph::view_t::csr /* | graph::view_t::csc */>(
          csr.number_of_rows,               // rows
          csr.number_of_columns,            // columns
          csr.number_of_nonzeros,           // nonzeros
          csr.row_offsets.data().get(),     // row_offsets
          csr.column_indices.data().get(),  // column_indices
          csr.nonzero_values.data().get(),  // values
          row_indices.data().get(),         // row_indices
          column_offsets.data().get()       // column_offsets
      );

  // --
  // Params and memory allocation

  vertex_t single_source = 0;

  vertex_t n_vertices = G.get_number_of_vertices();
  thrust::device_vector<vertex_t> distances(n_vertices);
  thrust::device_vector<vertex_t> predecessors(n_vertices);

  // --
  // Run problem

  float gpu_elapsed = gunrock::bfs::run(
      G, single_source, distances.data().get(), predecessors.data().get());

  // --
  // CPU Run

  thrust::host_vector<vertex_t> h_distances(n_vertices);
  thrust::host_vector<vertex_t> h_predecessors(n_vertices);

  float cpu_elapsed = bfs_cpu::run<csr_t, vertex_t, edge_t>(
      csr, single_source, h_distances.data(), h_predecessors.data());

  int n_errors =
      util::compare(distances.data().get(), h_distances.data(), n_vertices);

  // --
  // Log

  print::head(distances, 40, "GPU distances");
  print::head(h_distances, 40, "CPU Distances");

  std::cout << "GPU Elapsed Time : " << gpu_elapsed << " (ms)" << std::endl;
  std::cout << "CPU Elapsed Time : " << cpu_elapsed << " (ms)" << std::endl;
  std::cout << "Number of errors : " << n_errors << std::endl;
}

int main(int argc, char** argv) {
  test_bfs(argc, argv);
}
